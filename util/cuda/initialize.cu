#include <hip/hip_runtime.h>
#include "check.h"
#include "initialize.h"

static hipDeviceProp_t
current_device_prop()
{
  int device;
  CUDA_CHECK(hipGetDevice(&device));

  hipDeviceProp_t prop;
  CUDA_CHECK(hipGetDeviceProperties(&prop, device));

  return prop;
}

void
ispm_initialize()
{
  CUDA_CHECK(hipSetDeviceFlags(hipDeviceMapHost));
  CUDA_CHECK(hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte));
}

DeviceIDString
get_device_string()
{
  hipDeviceProp_t p = current_device_prop();

  DeviceIDString s;
  snprintf(s.str, sizeof(s.str), "%04x:%02x:%02x (%s)",
           p.pciDomainID, p.pciBusID, p.pciDeviceID, p.name);

  return s;
}
