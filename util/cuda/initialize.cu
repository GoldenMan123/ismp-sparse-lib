#include <hip/hip_runtime.h>
#include "check.h"
#include "initialize.h"

void
ispm_initialize()
{
  CUDA_CHECK(hipSetDeviceFlags(hipDeviceMapHost));
  CUDA_CHECK(hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte));
}

DeviceIDString
get_device_string()
{
  int device;
  CUDA_CHECK(hipGetDevice(&device));

  char busid[13];
  CUDA_CHECK(hipDeviceGetPCIBusId(busid, sizeof(busid), device));

  struct hipDeviceProp_t prop;
  CUDA_CHECK(hipGetDeviceProperties(&prop, device));

  DeviceIDString s;
  snprintf(s.str, sizeof(s.str), "%s (%s)", busid, prop.name);

  return s;
}
