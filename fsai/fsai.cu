#include "hip/hip_runtime.h"
#include "fsai/fsai.h"

#include <hip/hip_runtime_api.h>

#include <iostream>
#include <cmath>
#include <set>

extern double get_time();

template<typename real>
csr_matrix<real> csr_transpose(const csr_matrix<real> &A) {
    /// Init matrix
    csr_matrix<real> B;
    B.n_rows = A.n_cols;
    B.n_cols = A.n_rows;
    B.row_ptr.clear();
    B.row_ptr.resize(B.n_rows + 1);
    B.cols.clear();
    B.elms.clear();

    /// Allocate temp arrays
    int *rcount = new int[B.n_rows];
    int *rindex = new int[B.n_rows];

    for (int i = 0; i < B.n_rows; ++i) {
        rcount[i] = 0;
    }

    /// Calculate row count
    for (int i = 0; i < A.n_rows; ++i) {
        for (int j = A.row_ptr[i]; j < A.row_ptr[i + 1]; ++j) {
            ++rcount[A.cols[j]];
        }
    }

    /// Calculare row_ptr
    rindex[0] = 0;
    for (int i = 1; i < B.n_rows; ++i) {
        rindex[i] = rindex[i - 1] + rcount[i - 1];
    }

    for (int i = 0; i < B.n_rows; ++i) {
        B.row_ptr[i] = rindex[i];
    }
    B.n_nz = B.row_ptr[B.n_rows] = rindex[B.n_rows - 1] + rcount[B.n_rows - 1];

    /// Fill cols & elms
    B.cols.resize(B.n_nz);
    B.elms.resize(B.n_nz);

    for (int i = 0; i < A.n_rows; ++i) {
        for (int j = A.row_ptr[i]; j < A.row_ptr[i + 1]; ++j) {
            B.cols[rindex[A.cols[j]]] = i;
            B.elms[rindex[A.cols[j]]] = A.elms[j];
            ++rindex[A.cols[j]];
        }
    }

    /// Free temp arrays
    delete[] rcount;
    delete[] rindex;

    /// Return result
    return B;
}

template<typename in_real, typename out_real>
csr_matrix<out_real> generate_pattern(const csr_matrix<in_real> &A) {
    /// Init matrix
    csr_matrix<out_real> B;
    B.n_rows = A.n_rows;
    B.n_cols = A.n_cols;
    B.row_ptr.clear();
    B.row_ptr.resize(B.n_rows + 1);
    B.cols.clear();
    B.elms.clear();

    /// Allocate temp arrays
    int *rcount = new int[B.n_rows];
    int *rindex = new int[B.n_rows];

    for (int i = 0; i < B.n_rows; ++i) {
        rcount[i] = 1;
    }

    /// Calculate row count
    for (int i = 0; i < A.n_rows; ++i) {
        for (int j = A.row_ptr[i]; j < A.row_ptr[i + 1]; ++j) {
            if (A.cols[j] >= i) {
                break;
            }
            ++rcount[i];
        }
    }

    /// Calculare row_ptr
    rindex[0] = 0;
    for (int i = 1; i < B.n_rows; ++i) {
        rindex[i] = rindex[i - 1] + rcount[i - 1];
    }

    for (int i = 0; i < B.n_rows; ++i) {
        B.row_ptr[i] = rindex[i];
    }
    B.n_nz = B.row_ptr[B.n_rows] = rindex[B.n_rows - 1] + rcount[B.n_rows - 1];

    /// Allocate cols & elms
    B.cols.resize(B.n_nz);
    B.elms.resize(B.n_nz);

    /// Fill cols
    for (int i = 0; i < A.n_rows; ++i) {
        for (int j = A.row_ptr[i]; j < A.row_ptr[i + 1]; ++j) {
            if (A.cols[j] >= i) {
                break;
            }
            B.cols[rindex[i]] = A.cols[j];
            B.elms[rindex[i]] = 1;
            ++rindex[i];
        }
        B.cols[rindex[i]] = i;
        B.elms[rindex[i]] = 1;
    }

    /// Free temp arrays
    delete[] rcount;
    delete[] rindex;

    /// Return result
    return B;
}

template<typename real>
real real_abs(real x) {
    return x > 0 ? x : -x;
}

template<typename T>
void swap(T &a, T &b) {
    T c = a;
    a = b;
    b = c;
}

template<>
void swap(int &a, int &b) {
    a ^= b;
    b ^= a;
    a ^= b;
}

template<typename in_real, typename out_real>
vector<out_real> solve(dense_matrix<in_real> &A) {
    /// Define system size N
    int N = A.n_rows;

    /// Init X
    vector<out_real> X;
    X.resize(N);

    /// Init swaps array
    int *swp = new int[N];
    for (int i = 0; i < N; ++i) {
        swp[i] = i;
    }

    /// Gauss iterations
    for (int i = 0; i < N; ++i) {
        /// Find max element
        in_real max = real_abs(A.data[i][i]);
        int max_i = i;
        for (int j = i + 1; j < N; ++j) {
            if (real_abs(A.data[i][j]) > max) {
                max = real_abs(A.data[i][j]);
                max_i = j;
            }
        }

        /// Swap cols
        if (i != max_i) {
            swap(swp[i], swp[max_i]);
            for (int j = 0; j < N; ++j) {
                swap(A.data[j][i], A.data[j][max_i]);
            }
        }

        /// Subtract rows
        in_real tmp = A.data[i][i];
        for (int j = i; j < N; ++j) {
            A.data[i][j] /= tmp;
        }
        A.data[i][N] /= tmp;
        for (int j = i + 1; j < N; ++j) {
            tmp = A.data[j][i];
            for (int k = i; k <= N; ++k) {
                A.data[j][k] -= tmp * A.data[i][k];
            }
        }
    }

    /// Calculate answer
    for (int i = N; i > 0; --i) {
        X[swp[i - 1]] = A.data[i - 1][N];
        for (int j = i; j < N; ++j) {
            X[swp[i - 1]] -= A.data[i - 1][j] * X[swp[j]];
        }
    }

    /// Free temp array
    delete[] swp;

    /// Return X
    return X;
}

template<typename in_real, typename out_real>
void calculate_factor(csr_matrix<out_real> &F, const csr_matrix<in_real> &A) {
    /// Calculate F's rows
    for (int row = 0; row < F.n_rows; ++row) {
        /// Init matrix for small system
        dense_matrix<in_real> S;
        S.n_rows = F.row_ptr[row + 1] - F.row_ptr[row];
        S.n_cols = S.n_rows + 1;
        S.data.resize(S.n_rows);
        for (int i = 0; i < S.n_rows; ++i) {
            S.data[i].resize(S.n_cols);
        }

        /// Generate small system
        for (int i = 0; i < S.n_rows; ++i) {
            int a_row = F.cols[F.row_ptr[row] + i];
            int f1 = F.row_ptr[row];
            int f1e = F.row_ptr[row + 1];
            int f2 = A.row_ptr[a_row];
            int f2e = A.row_ptr[a_row + 1];
            int j = 0;
            while ((f1 < f1e) && (f2 < f2e)) {
                if (F.cols[f1] == A.cols[f2]) {
                    S.data[i][j] = A.elms[f2];
                    ++f1;
                    ++f2;
                    ++j;
                } else {
                    if (F.cols[f1] < A.cols[f2]) {
                        S.data[i][j] = 0;
                        ++f1;
                        ++j;
                    } else {
                        ++f2;
                    }
                }
            }
            while (j < S.n_cols - 1) {
                S.data[i][j] = 0;
                ++j;
            }
            S.data[i][S.n_cols - 1] = ((i == S.n_rows - 1) ? 1 : 0);
        }

        /// Solve small system
        vector<out_real> X = solve<in_real, out_real>(S);
        if (abs(X.back()) > 1e-8) {
            double dividor = sqrt(abs(X.back()));
            for (int i = 0; i < X.size(); ++i) {
                X[i] /= dividor;
            }
        }
        for (int i = 0; i < X.size(); ++i) {
            F.elms[F.row_ptr[row] + i] = X[i];
        }
    }
}

template<typename real>
csr_matrix<real> out_transform(const csr_matrix<real> &A) {
    /// Init matrix
    csr_matrix<real> B;
    B.n_rows = A.n_rows;
    B.n_cols = A.n_cols;
    B.row_ptr.clear();
    B.row_ptr.resize(B.n_rows + 1);
    B.cols.clear();
    B.elms.clear();

    /// Allocate temp arrays
    int *rcount = new int[B.n_rows];
    int *rindex = new int[B.n_rows];

    for (int i = 0; i < A.n_rows; ++i) {
        rcount[i] = 0;
    }

    /// Calculate row count
    for (int i = 0; i < A.n_rows; ++i) {
        for (int j = A.row_ptr[i]; j < A.row_ptr[i + 1]; ++j) {
            ++rcount[A.n_rows - 1 - i];
        }
    }

    /// Calculare row_ptr
    rindex[0] = 0;
    for (int i = 1; i < B.n_rows; ++i) {
        rindex[i] = rindex[i - 1] + rcount[i - 1];
    }

    for (int i = 0; i < B.n_rows; ++i) {
        B.row_ptr[i] = rindex[i];
    }
    B.n_nz = B.row_ptr[B.n_rows] = rindex[B.n_rows - 1] + rcount[B.n_rows - 1];

    /// Fill cols & elms
    B.cols.resize(B.n_nz);
    B.elms.resize(B.n_nz);

    for (int i = 0; i < A.n_rows; ++i) {
        for (int j = A.row_ptr[i]; j < A.row_ptr[i + 1]; ++j) {
            B.cols[rindex[A.n_rows - 1 - i]] = A.cols[j];
            B.elms[rindex[A.n_rows - 1 - i]] = A.elms[j];
            ++rindex[A.n_rows - 1 - i];
        }
    }

    /// Free temp arrays
    delete[] rcount;
    delete[] rindex;

    /// Return result
    return B;
}

template<typename in_real, typename out_real>
csr_matrix<out_real> square_pattern(const csr_matrix<in_real> &A) {
    /// Init matrix
    csr_matrix<out_real> B;
    B.n_rows = A.n_rows;
    B.n_cols = A.n_cols;
    B.row_ptr.clear();
    B.row_ptr.resize(B.n_rows + 1);
    B.cols.clear();
    B.elms.clear();

    /// Allocate temp arrays
    int *rcount = new int[B.n_rows];
    int *rindex = new int[B.n_rows];

    /// Calculate row count
    for (int i = 0; i < B.n_rows; ++i) {
        std::set<int> tmp;
        tmp.insert(i);
        for (int j = A.row_ptr[i]; j < A.row_ptr[i + 1]; ++j) {
            for (int k = A.row_ptr[A.cols[j]]; k < A.row_ptr[A.cols[j] + 1]; ++k) {
                if (A.cols[k] >= i) {
                    break;
                }
                tmp.insert(A.cols[k]);
            }
        }
        rcount[i] = tmp.size();
    }

    /// Calculare row_ptr
    rindex[0] = 0;
    for (int i = 1; i < B.n_rows; ++i) {
        rindex[i] = rindex[i - 1] + rcount[i - 1];
    }

    for (int i = 0; i < B.n_rows; ++i) {
        B.row_ptr[i] = rindex[i];
    }
    B.n_nz = B.row_ptr[B.n_rows] = rindex[B.n_rows - 1] + rcount[B.n_rows - 1];

    /// Fill cols & elms
    B.cols.resize(B.n_nz);
    B.elms.resize(B.n_nz);

    for (int i = 0; i < B.n_rows; ++i) {
        std::set<int> tmp;
        tmp.insert(i);
        for (int j = A.row_ptr[i]; j < A.row_ptr[i + 1]; ++j) {
            for (int k = A.row_ptr[A.cols[j]]; k < A.row_ptr[A.cols[j] + 1]; ++k) {
                if (A.cols[k] >= i) {
                    break;
                }
                tmp.insert(A.cols[k]);
            }
        }
        for (std::set<int>::iterator j = tmp.begin(); j != tmp.end(); ++j) {
            B.cols[rindex[i]] = *j;
            B.elms[rindex[i]] = 1;
            ++rindex[i];
        }
    }

    /// Free temp arrays
    delete[] rcount;
    delete[] rindex;

    /// Return result
    return B;
}

static double PROFILE_TIME;

static void PROFILE_BEGIN() {
    PROFILE_TIME = get_time();
}

static void PROFILE_END(const std::string &prefix) {
    std::cerr << prefix << ": " << std::setprecision(3) << std::fixed << 1000.0 * (get_time() - PROFILE_TIME) << "ms" << std::endl;
}

template<typename in_real, typename out_real>
void fsai_impl(csr_matrix<out_real> &Ainv1,
    csr_matrix<out_real> &Ainv2,
    const csr_matrix<in_real> &A) {

    /// Calculate transposed A
    PROFILE_BEGIN();
    csr_matrix<in_real> AT = csr_transpose(A);
    PROFILE_END("csr_transpose");

    /// Generate pattern for Ainv1
    PROFILE_BEGIN();
    Ainv1 = square_pattern<in_real, out_real>(A);
    PROFILE_END("square_pattern");

    /// Calculate Ainv1
    PROFILE_BEGIN();
    calculate_factor(Ainv1, AT);
    PROFILE_END("calculate_factor");

    /// Transform Ainv1 (for CUDA solver)
    //Ainv1 = out_transform(Ainv1);

    /// Calculate transposed Ainv2
    PROFILE_BEGIN();
    Ainv2 = csr_transpose(Ainv1);
    PROFILE_END("csr_transpose");

}

void fsai(csr_matrix<float> &Ainv1,
    csr_matrix<float> &Ainv2,
    const csr_matrix<double> &A) {
    fsai_impl<double, float>(Ainv1, Ainv2, A);
}

void fsai(csr_matrix<double> &Ainv1,
    csr_matrix<double> &Ainv2,
    const csr_matrix<double> &A) {
    fsai_impl<double, double>(Ainv1, Ainv2, A);
}
